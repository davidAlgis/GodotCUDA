#include "hip/hip_runtime.h"
#include "cuda_example.h"
#include <hip/hip_runtime.h>

namespace cudaTest
{
/**
 * @brief A kernel function that performs a simple computation on an array of
 * values.
 *
 * @param input The input array containing values to compute.
 * @param output The output array to store the results.
 */
__global__ void testkernel(const int *input, float *output)
{
    *output = static_cast<float>(*input) * 2.5f;
}

float doCalculation(int inputValue)
{
    int *d_input;
    float *d_output;
    int h_input[1] = {inputValue};
    float h_output[1] = {1.0f};

    // Allocate memory on the device
    hipMalloc(&d_input, sizeof(int));
    hipMalloc(&d_output, sizeof(float));
    hipMemcpy(d_input, h_input, sizeof(int), hipMemcpyHostToDevice);

    // Launch the kernel
    testkernel<<<1, 1>>>(d_input, d_output);

    // Copy the results back to the host
    hipMemcpy(h_output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
    float result = h_output[0];
    return result;
}
} // namespace cudaTest