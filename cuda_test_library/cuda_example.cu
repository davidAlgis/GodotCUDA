#include "hip/hip_runtime.h"
#include "cuda_example.h"
#include <hip/hip_runtime.h>

namespace cudaTest
{
/**
 * @brief A kernel function that performs a heavy computation on an array of
 * values.
 *
 * This kernel squares each element in the input array and stores the result in
 * the output array. Each thread is responsible for computing the square of a
 * single element, allowing for parallel computation across multiple threads.
 *
 * @param input The input array containing values to compute.
 * @param output The output array to store the results.
 * @param size The number of elements in the input/output arrays.
 */
__global__ void computeHeavyKernel(const int *input, float *output, int size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
    {
        output[idx] =
            static_cast<float>(input[idx]) * input[idx]; // Square the value
    }
}

/**
 * @brief A function that performs a heavy calculation using CUDA.
 *
 * This function takes an integer as input and initializes a large array with
 * this value, then uses CUDA to compute the square of each element in the array
 * in parallel. The results are copied back to the host, and the sum of all
 * elements in the output array is returned as the result.
 *
 * @param inputValue The integer value used to populate the input array.
 * @return The sum of all squared values in the output array.
 */
float doCalculation(int inputValue)
{
    const int arraySize = 1024;
    int hostInput[arraySize];
    float hostOutput[arraySize] = {0};
    int *deviceInput;
    float *deviceOutput;

    // Initialize the input array on the host
    for (int i = 0; i < arraySize; ++i)
    {
        hostInput[i] = inputValue;
    }

    // Allocate memory on the device
    hipMalloc(&deviceInput, arraySize * sizeof(int));
    hipMalloc(&deviceOutput, arraySize * sizeof(float));

    // Copy the host input array to the device
    hipMemcpy(deviceInput, hostInput, arraySize * sizeof(int),
               hipMemcpyHostToDevice);

    // Launch the kernel with enough threads to cover all elements
    int threadsPerBlock = 256;
    int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;
    computeHeavyKernel<<<blocksPerGrid, threadsPerBlock>>>(
        deviceInput, deviceOutput, arraySize);

    // Copy the results back to the host
    hipMemcpy(hostOutput, deviceOutput, arraySize * sizeof(float),
               hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(deviceInput);
    hipFree(deviceOutput);

    // Sum all results
    float result = 0.0f;
    for (int i = 0; i < arraySize; ++i)
    {
        result += hostOutput[i];
    }

    return result;
}
} // namespace cudaTest