#include "hip/hip_runtime.h"
#include "cuda_example.h"
#include <hip/hip_runtime.h>

__global__ void computeKernel(float *result)
{
    int idx = threadIdx.x;
    result[idx] = idx * 2.0f; // Simple computation
}

extern "C" float doCalculation()
{
    const int arraySize = 1;
    float hostResult[arraySize] = {0};
    float *deviceResult;

    hipMalloc((void **)&deviceResult, arraySize * sizeof(float));
    computeKernel<<<1, arraySize>>>(deviceResult);
    hipMemcpy(hostResult, deviceResult, arraySize * sizeof(float),
               hipMemcpyDeviceToHost);
    hipFree(deviceResult);

    return hostResult[0];
}
